
// CUDA运行时头文件
#include <hip/hip_runtime.h>

#include <chrono>
#include <stdio.h>
#include <string.h>

using namespace std;

#define checkRuntime(op)  __check_cuda_runtime((op), #op, __FILE__, __LINE__)

bool __check_cuda_runtime(hipError_t code, const char* op, const char* file, int line){
    if(code != hipSuccess){    
        const char* err_name = hipGetErrorName(code);    
        const char* err_message = hipGetErrorString(code);  
        printf("runtime error %s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_message);   
        return false;
    }
    return true;
}

__global__ void add_vector(const float* a, const float* b, float* c, int count){

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index >= count) return;
    c[index] = a[index] + b[index];
}

__global__ void mul_vector(const float* a, const float* b, float* c, int count){

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index >= count) return;
    c[index] = a[index] * b[index];
}

hipStream_t stream1, stream2;
float *a, *b, *c1, *c2;
const int num_element = 100000;
const size_t bytes = sizeof(float) * num_element;
const int blocks = 512;
const int grids = (num_element + blocks - 1) / blocks;
const int ntry = 1000;

// 多个流异步
void async(){

    hipEvent_t event_start1, event_stop1;
    hipEvent_t event_start2, event_stop2;
    checkRuntime(hipEventCreate(&event_start1));
    checkRuntime(hipEventCreate(&event_stop1));
    checkRuntime(hipEventCreate(&event_start2));
    checkRuntime(hipEventCreate(&event_stop2));

    auto tic = chrono::duration_cast<chrono::microseconds>(chrono::system_clock::now().time_since_epoch()).count() / 1000.0;
    checkRuntime(hipEventRecord(event_start1, stream1));
    for(int i = 0; i < ntry; ++i)
        add_vector<<<grids, blocks, 0, stream1>>>(a, b, c1, num_element);
    checkRuntime(hipEventRecord(event_stop1, stream1));
    
    checkRuntime(hipEventRecord(event_start2, stream2));
    for(int i = 0; i < ntry; ++i)
        add_vector<<<grids, blocks, 0, stream2>>>(a, b, c2, num_element);
    checkRuntime(hipEventRecord(event_stop2, stream2));

    checkRuntime(hipStreamSynchronize(stream1));
    checkRuntime(hipStreamSynchronize(stream2));
    auto toc = chrono::duration_cast<chrono::microseconds>(chrono::system_clock::now().time_since_epoch()).count() / 1000.0;

    float time1, time2;
    checkRuntime(hipEventElapsedTime(&time1, event_start1, event_stop1));
    checkRuntime(hipEventElapsedTime(&time2, event_start2, event_stop2));
    printf("async: time1 = %.2f ms, time2 = %.2f ms, count = %.2f ms\n", time1, time2, toc - tic);
}

// 单个流串行
void sync(){

    hipEvent_t event_start1, event_stop1;
    checkRuntime(hipEventCreate(&event_start1));
    checkRuntime(hipEventCreate(&event_stop1));

    auto tic = chrono::duration_cast<chrono::microseconds>(chrono::system_clock::now().time_since_epoch()).count() / 1000.0;
    checkRuntime(hipEventRecord(event_start1, stream1));
    for(int i = 0; i < ntry; ++i)
        add_vector<<<grids, blocks, 0, stream1>>>(a, b, c1, num_element);
    
    for(int i = 0; i < ntry; ++i)
        add_vector<<<grids, blocks, 0, stream1>>>(a, b, c2, num_element);
    checkRuntime(hipEventRecord(event_stop1, stream1));

    checkRuntime(hipStreamSynchronize(stream1));
    auto toc = chrono::duration_cast<chrono::microseconds>(chrono::system_clock::now().time_since_epoch()).count() / 1000.0;

    float time1;
    checkRuntime(hipEventElapsedTime(&time1, event_start1, event_stop1));
    printf("sync: time1 = %.2f ms, count = %.2f ms\n", time1, toc - tic);
}

// 多个流之间并行
void multi_stream_async(){

    // 这个案例主要实现多个流之间互相等待，使用event控制实现
    // 存在step1  ->  step2 \ 
    //                      ->  step3   ->  step4
    //               stepa / 
    //
    // 这个案例中，存在流程1：step1 -> step2的流程
    //           存在流程2：stepa
    //           存在流程3：step3 -> step4，step3要求step2与stepa作为输入
    // 此时，可以让流程1使用stream1，流程2使用stream2，而流程3继续使用stream1，仅仅在stream1中加入等待（event的等待）

    // step1 = add_vector
    // step2 = mul_vector
    // step3 = add_vector
    // step4 = mul_vector
    // stepa = add_vector
    #define step1 add_vector
    #define step2 mul_vector
    #define step3 add_vector
    #define step4 mul_vector
    #define stepa add_vector

    hipEvent_t event_async;
    checkRuntime(hipEventCreate(&event_async));

    // stream1的执行流程
    step1<<<grids, blocks, 0, stream1>>>(a, b, c1, num_element);
    step2<<<grids, blocks, 0, stream1>>>(a, b, c1, num_element);

    // 等待event_async有事件
    checkRuntime(hipStreamWaitEvent(stream1, event_async));
    step3<<<grids, blocks, 0, stream1>>>(a, b, c2, num_element);
    step4<<<grids, blocks, 0, stream1>>>(a, b, c2, num_element);

    // stream2的执行流程
    stepa<<<grids, blocks, 0, stream2>>>(a, b, c2, num_element);
    
    // 为event_async触发事件，通知cudaStreamWaitEvent函数可以继续了
    checkRuntime(hipEventRecord(event_async, stream2));
    checkRuntime(hipStreamSynchronize(stream1));

    printf("multi_stream_async done.\n");
}

int main(){

    // 本程序实现两个核函数的并行，通过多个流实现
    
    checkRuntime(hipStreamCreate(&stream1));
    checkRuntime(hipStreamCreate(&stream2));

    checkRuntime(hipMalloc(&a, bytes));
    checkRuntime(hipMalloc(&b, bytes));
    checkRuntime(hipMalloc(&c1, bytes));
    checkRuntime(hipMalloc(&c2, bytes));

    // 演示多流之间的异步执行
    async();

    // 演示单个流内的同步执行
    sync();

    // 演示多个流之间互相等待的操作
    multi_stream_async();
    return 0;
}